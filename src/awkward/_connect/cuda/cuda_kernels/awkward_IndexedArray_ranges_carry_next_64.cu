#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (index, fromstarts, fromstops, length, tocarry, invocation_index, err_code) = args
//     scan_in_array = cupy.zeros_like(index, dtype=cupy.int64)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_IndexedArray_ranges_carry_next_64_a", index.dtype, fromstarts.dtype, fromstops.dtype, tocarry.dtype]))(grid, block, (index, fromstarts, fromstops, length, tocarry, scan_in_array, invocation_index, err_code))
//     scan_in_array = cupy.cumsum(scan_in_array)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_IndexedArray_ranges_carry_next_64_b", index.dtype, fromstarts.dtype, fromstops.dtype, tocarry.dtype]))(grid, block, (index, fromstarts, fromstops, length, tocarry, scan_in_array, invocation_index, err_code))
// out["awkward_IndexedArray_ranges_carry_next_64_a", {dtype_specializations}] = None
// out["awkward_IndexedArray_ranges_carry_next_64_b", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C, typename U, typename V>
__global__ void
awkward_IndexedArray_ranges_carry_next_64_a(
    T* index,
    const C* fromstarts,
    const U* fromstops,
    int64_t length,
    V* tocarry,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t stride = 0;

    if (thread_id < length) {
      stride = fromstops[thread_id] - fromstarts[thread_id];
      for (int64_t j = threadIdx.y; j < stride; j += blockDim.y) {
        if (!(index[fromstarts[thread_id] + j] < 0)) {
          scan_in_array[fromstarts[thread_id] + j] = 1;
        }
      }
    }
  }
}

template <typename T, typename C, typename U, typename V>
__global__ void
awkward_IndexedArray_ranges_carry_next_64_b(
    T* index,
    const C* fromstarts,
    const U* fromstops,
    int64_t length,
    V* tocarry,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t stride = 0;

    if (thread_id < length) {
      stride = fromstops[thread_id] - fromstarts[thread_id];
      for (int64_t j = threadIdx.y; j < stride; j += blockDim.y) {
        if (!(index[fromstarts[thread_id] + j] < 0)) {
          tocarry[scan_in_array[fromstarts[thread_id] + j] - 1] = index[fromstarts[thread_id] + j];
        }
      }
    }
  }
}
