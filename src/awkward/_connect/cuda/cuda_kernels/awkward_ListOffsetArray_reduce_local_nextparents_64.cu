#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T, typename C>
__global__ void
awkward_ListOffsetArray_reduce_local_nextparents_64(
    T* nextparents,
    const C* offsets,
    int64_t length,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < length) {
      int64_t initialoffset = (int64_t)(offsets[0]);
      int64_t start = (int64_t)(offsets[thread_id]) - initialoffset;
      int64_t stop = (int64_t)offsets[thread_id + 1] - initialoffset;
      for (int64_t j = start + threadIdx.y; j < stop; j += blockDim.y) {
        nextparents[j] = thread_id;
      }
    }
  }
}
