#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (toptr, fromptr, parents, lenparents, outlength, invocation_index, err_code) = args
//     if block[0] > 0:
//         grid_size = math.floor((lenparents + block[0] - 1) / block[0])
//     else:
//         grid_size = 1
//     atomic_toptr = cupy.array(toptr, dtype=toptr.dtype)
//     temp = cupy.ones(lenparents, dtype=toptr.dtype)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_prod_a", cupy.dtype(toptr.dtype).type, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, atomic_toptr, temp, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_prod_b", cupy.dtype(toptr.dtype).type, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, atomic_toptr, temp, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_prod_c", cupy.dtype(toptr.dtype).type, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, atomic_toptr, temp, invocation_index, err_code))
// out["awkward_reduce_prod_a", {dtype_specializations}] = None
// out["awkward_reduce_prod_b", {dtype_specializations}] = None
// out["awkward_reduce_prod_c", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_prod_a(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    T* atomic_toptr,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < outlength) {
      atomic_toptr[thread_id] = 1;
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_prod_b(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    T* atomic_toptr,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t idx = threadIdx.x;
    int64_t thread_id = blockIdx.x * blockDim.x + idx;

    if (thread_id < lenparents) {
      temp[thread_id] = fromptr[thread_id];
    }
    __syncthreads();

    if (thread_id < lenparents) {
      for (int64_t stride = 1; stride < blockDim.x; stride *= 2) {
        T val = 1;
        if (idx >= stride && thread_id < lenparents && parents[thread_id] == parents[thread_id - stride]) {
          val = temp[thread_id - stride];
        }
        __syncthreads();
        temp[thread_id] *= val;
        __syncthreads();
      }

      int64_t parent = parents[thread_id];
      if (idx == blockDim.x - 1 || thread_id == lenparents - 1 || parents[thread_id] != parents[thread_id + 1]) {
        atomicMul(&atomic_toptr[parent], temp[thread_id]);
      }
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_prod_c(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    T* atomic_toptr,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < outlength) {
      toptr[thread_id] = (T)(atomic_toptr[thread_id]);
    }
  }
}
