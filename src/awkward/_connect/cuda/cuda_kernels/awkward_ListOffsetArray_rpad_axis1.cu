#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (toindex, fromoffsets, fromlength, target, invocation_index, err_code) = args
//     scan_in_array = cupy.zeros(fromlength, dtype=cupy.int64)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_ListOffsetArray_rpad_axis1_a", toindex.dtype, fromoffsets.dtype]))(grid, block, (toindex, fromoffsets, fromlength, target, scan_in_array, invocation_index, err_code))
//     scan_in_array = cupy.cumsum(scan_in_array)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_ListOffsetArray_rpad_axis1_b", toindex.dtype, fromoffsets.dtype]))(grid, block, (toindex, fromoffsets, fromlength, target, scan_in_array, invocation_index, err_code))
// out["awkward_ListOffsetArray_rpad_axis1_a", {dtype_specializations}] = None
// out["awkward_ListOffsetArray_rpad_axis1_b", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C>
__global__ void
awkward_ListOffsetArray_rpad_axis1_a(
    T* toindex,
    const C* fromoffsets,
    int64_t fromlength,
    int64_t target,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < fromlength) {
      int64_t rangeval =
          (T)(fromoffsets[thread_id + 1] - fromoffsets[thread_id]);
      scan_in_array[thread_id + 1] = rangeval > target ? rangeval : target;
    }
  }
}

template <typename T, typename C>
__global__ void
awkward_ListOffsetArray_rpad_axis1_b(
    T* toindex,
    const C* fromoffsets,
    int64_t fromlength,
    int64_t target,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < fromlength) {
      int64_t rangeval =
          (T)(fromoffsets[thread_id + 1] - fromoffsets[thread_id]);
      int64_t index = scan_in_array[thread_id];

      for (int64_t j = threadIdx.y; j < rangeval; j += blockDim.y) {
        toindex[index + j] = (T)fromoffsets[thread_id] + j;
      }
      for (int64_t j = rangeval + threadIdx.y; j < target; j += blockDim.y) {
        toindex[index + j] = -1;
      }
    }
  }
}
