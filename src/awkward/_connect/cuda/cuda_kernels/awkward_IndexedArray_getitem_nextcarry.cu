#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

enum class INDEXEDARRAY_GETITEM_NEXTCARRY_ERRORS {
  IND_OUT_OF_RANGE,  // message: "index out of range"
};

// BEGIN PYTHON
// def f(grid, block, args):
//     (tocarry, fromindex, lenindex, lencontent, invocation_index, err_code) = args
//     scan_in_array = cupy.zeros(lenindex, dtype=cupy.int64)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_IndexedArray_getitem_nextcarry_a", tocarry.dtype, fromindex.dtype]))(grid, block, (tocarry, fromindex, lenindex, lencontent, scan_in_array, invocation_index, err_code))
//     scan_in_array = cupy.cumsum(scan_in_array)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_IndexedArray_getitem_nextcarry_b", tocarry.dtype, fromindex.dtype]))(grid, block, (tocarry, fromindex, lenindex, lencontent, scan_in_array, invocation_index, err_code))
// out["awkward_IndexedArray_getitem_nextcarry_a", {dtype_specializations}] = None
// out["awkward_IndexedArray_getitem_nextcarry_b", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C>
__global__ void
awkward_IndexedArray_getitem_nextcarry_a(
    T* tocarry,
    const C* fromindex,
    int64_t lenindex,
    int64_t lencontent,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < lenindex) {
      C j = fromindex[thread_id];
      if (j < 0 || j >= lencontent) {
        RAISE_ERROR(INDEXEDARRAY_GETITEM_NEXTCARRY_ERRORS::IND_OUT_OF_RANGE)
      } else {
        scan_in_array[thread_id] = 1;
      }
    }
  }
}

template <typename T, typename C>
__global__ void
awkward_IndexedArray_getitem_nextcarry_b(
    T* tocarry,
    const C* fromindex,
    int64_t lenindex,
    int64_t lencontent,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < lenindex) {
      C j = fromindex[thread_id];
      if (j < 0 || j >= lencontent) {
        RAISE_ERROR(INDEXEDARRAY_GETITEM_NEXTCARRY_ERRORS::IND_OUT_OF_RANGE)
      } else {
        tocarry[scan_in_array[thread_id] - 1] = j;
      }
    }
  }
}
