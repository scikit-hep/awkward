#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (toptr, fromptr, parents, lenparents, outlength, invocation_index, err_code) = args
//     if block[0] > 0:
//         grid_size = math.floor((lenparents + block[0] - 1) / block[0])
//     else:
//         grid_size = 1
//     temp = cupy.zeros(2 * lenparents, dtype=toptr.dtype)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_sum_complex_a", cupy.dtype(toptr.dtype).type, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, temp, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_sum_complex_b", cupy.dtype(toptr.dtype).type, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, temp, invocation_index, err_code))
// out["awkward_reduce_sum_complex_a", {dtype_specializations}] = None
// out["awkward_reduce_sum_complex_b", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_sum_complex_a(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < outlength) {
      toptr[thread_id * 2] = (T)0;
      toptr[thread_id * 2 + 1] = (T)0;
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_sum_complex_b(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t idx = threadIdx.x;
    int64_t thread_id = blockIdx.x * blockDim.x + idx;

    if (thread_id < lenparents) {
      temp[thread_id * 2] = fromptr[thread_id * 2];
      temp[thread_id * 2 + 1] = fromptr[thread_id * 2 + 1];
    }
    __syncthreads();

    for (int64_t stride = 1; stride < blockDim.x; stride *= 2) {
      T real = (T)0;
      T imag = (T)0;
      if (idx >= stride && thread_id < lenparents && parents[thread_id] == parents[thread_id - stride]) {
        real = temp[(thread_id - stride) * 2];
        imag = temp[(thread_id - stride) * 2 + 1];
      }
      __syncthreads();
      temp[thread_id * 2] += real;
      temp[thread_id * 2 + 1] += imag;
      __syncthreads();
    }

    if (thread_id < lenparents) {
      int64_t parent = parents[thread_id];
      if (idx == blockDim.x - 1 || thread_id == lenparents - 1 || parents[thread_id] != parents[thread_id + 1]) {
        atomicAdd(&toptr[parent * 2], temp[thread_id * 2]);
        atomicAdd(&toptr[parent * 2 + 1], temp[thread_id * 2 + 1]);
      }
    }
  }
}
