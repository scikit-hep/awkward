#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T, typename C>
__global__ void
awkward_ListArray_localindex(
    T* toindex,
    const C* offsets,
    int64_t length,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      int64_t start = (int64_t)offsets[thread_id];
      int64_t stop = (int64_t)offsets[thread_id + 1];
      for (int64_t j = start + threadIdx.y;  j < stop;  j += blockDim.y) {
        toindex[j] = j - start;
      }
    }
  }
}
