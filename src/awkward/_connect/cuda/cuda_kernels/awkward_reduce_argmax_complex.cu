#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (toptr, fromptr, parents, lenparents, outlength, invocation_index, err_code) = args
//     if block[0] > 0:
//         grid_size = math.floor((lenparents + block[0] - 1) / block[0])
//     else:
//         grid_size = 1
//     atomic_toptr = cupy.array(toptr, dtype=cupy.uint64)
//     temp = cupy.zeros(lenparents, dtype=toptr.dtype)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_argmax_complex_a", cupy.dtype(toptr.dtype).type, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, atomic_toptr, temp, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_argmax_complex_b", cupy.dtype(toptr.dtype).type, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, atomic_toptr, temp, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_argmax_complex_c", cupy.dtype(toptr.dtype).type, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, atomic_toptr, temp, invocation_index, err_code))
// out["awkward_reduce_argmax_complex_a", {dtype_specializations}] = None
// out["awkward_reduce_argmax_complex_b", {dtype_specializations}] = None
// out["awkward_reduce_argmax_complex_c", {dtype_specializations}] = None
// END PYTHON

// argmax: lexicographic compare (real, then imag), tie-breaker: lower index
template <typename T, typename C, typename U>
__global__ void
awkward_reduce_argmax_complex_a(
    T* toptr,
    const C* fromptr,   // [real, imag] pairs
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint64_t* atomic_toptr,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < outlength) {
      atomic_toptr[thread_id] = (uint64_t)(-1);
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_argmax_complex_b(
    T* toptr,
    const C* fromptr,   // flattened [real, imag] array
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint64_t* atomic_toptr,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {

  if (err_code[0] == NO_ERROR) {
    const uint64_t EMPTY = (uint64_t)(-1);

    int64_t idx = threadIdx.x;
    int64_t thread_id = blockIdx.x * blockDim.x + idx;

    if (thread_id < lenparents) {
      temp[thread_id] = thread_id;
    } else if (thread_id < outlength) {
      temp[thread_id] = -1;
    }
    __syncthreads();

    if (thread_id < lenparents) {
      // intra-block reduction
      for (int64_t stride = 1; stride < blockDim.x; stride *= 2) {
        int64_t index = -1;
        if (idx >= stride && parents[thread_id] == parents[thread_id - stride]) {
          index = temp[thread_id - stride];
        }
        __syncthreads();

        if (index != -1) {
          // load old candidate
          int64_t old_idx = temp[thread_id];
          double old_real = (double)fromptr[2 * old_idx];
          double old_imag = (double)fromptr[2 * old_idx + 1];
          // load new candidate
          double new_real = (double)fromptr[2 * index];
          double new_imag = (double)fromptr[2 * index + 1];

          // NUMPY LEXICOGRAPHIC (real, then imag), prefer lower index on ties
          bool better = (new_real > old_real) ||
                        (new_real == old_real && new_imag > old_imag) ||
                        (new_real == old_real && new_imag == old_imag && index < old_idx);

          if (old_idx == -1 || better) {
            temp[thread_id] = index;
          }
        }
        __syncthreads();
      }

      int64_t parent = parents[thread_id];
      if (idx == blockDim.x - 1 || thread_id == lenparents - 1 ||
          parents[thread_id] != parents[thread_id + 1]) {
        uint64_t candidate = (uint64_t)temp[thread_id];
        if (candidate != (uint64_t)-1) {
          uint64_t cur = atomic_toptr[parent];
          while (true) {
            if (cur == EMPTY) {
              uint64_t prev = atomicCAS(&atomic_toptr[parent], EMPTY, candidate);
              if (prev == EMPTY) break;
              cur = prev;
              continue;
            } else {
              int64_t old_idx = (int64_t)cur;
              int64_t new_idx = (int64_t)candidate;

              double old_real = (double)fromptr[2 * old_idx];
              double old_imag = (double)fromptr[2 * old_idx + 1];
              double new_real = (double)fromptr[2 * new_idx];
              double new_imag = (double)fromptr[2 * new_idx + 1];

              bool better = (new_real > old_real) ||
                            (new_real == old_real && new_imag > old_imag) ||
                            (new_real == old_real && new_imag == old_imag && new_idx < old_idx);

              if (better) {
                uint64_t prev = atomicCAS(&atomic_toptr[parent], cur, candidate);
                if (prev == cur) break;
                cur = prev;
                continue;
              } else {
                break;
              }
            }
          }
        }
      }
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_argmax_complex_c(
    T* toptr,
    const C* fromptr,   // unused
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint64_t* atomic_toptr,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < outlength) {
      toptr[thread_id] = (T)(atomic_toptr[thread_id]);
    }
  }
}
