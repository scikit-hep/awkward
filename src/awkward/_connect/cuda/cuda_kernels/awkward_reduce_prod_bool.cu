#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (toptr, fromptr, parents, lenparents, outlength, invocation_index, err_code) = args
//     if block[0] > 0:
//         grid_size = math.floor((lenparents + block[0] - 1) / block[0])
//     else:
//         grid_size = 1
//     atomic_toptr = cupy.array(toptr, dtype=cupy.uint32)
//     temp = cupy.ones(lenparents, dtype=toptr.dtype)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_prod_bool_a", bool_, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, atomic_toptr, temp, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_prod_bool_b", bool_, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, atomic_toptr, temp, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_prod_bool_c", bool_, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, atomic_toptr, temp, invocation_index, err_code))
// out["awkward_reduce_prod_bool_a", {dtype_specializations}] = None
// out["awkward_reduce_prod_bool_b", {dtype_specializations}] = None
// out["awkward_reduce_prod_bool_c", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_prod_bool_a(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint32_t* atomic_toptr,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < outlength) {
      atomic_toptr[thread_id] = 1;
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_prod_bool_b(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint32_t* atomic_toptr,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t idx = threadIdx.x;
    int64_t thread_id = blockIdx.x * blockDim.x + idx;

    if (thread_id < lenparents) {
      temp[thread_id] = fromptr[thread_id];
    }
    __syncthreads();

    if (thread_id < lenparents) {
      for (int64_t stride = 1; stride < blockDim.x; stride *= 2) {
        T val = 1;
        if (idx >= stride && thread_id < lenparents && parents[thread_id] == parents[thread_id - stride]) {
          val = temp[thread_id - stride];
        }
        __syncthreads();
        temp[thread_id] &= (val != 0);
        __syncthreads();
      }

      int64_t parent = parents[thread_id];
      if (idx == blockDim.x - 1 || thread_id == lenparents - 1 || parents[thread_id] != parents[thread_id + 1]) {
        atomicAnd(&atomic_toptr[parent], temp[thread_id]);
      }
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_prod_bool_c(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint32_t* atomic_toptr,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < outlength) {
      toptr[thread_id] = (T)(atomic_toptr[thread_id]);
    }
  }
}
