#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (toptr, fromptr, parents, lenparents, outlength, invocation_index, err_code) = args
//     if block[0] > 0:
//         grid_size = math.floor((lenparents + block[0] - 1) / block[0])
//     else:
//         grid_size = 1
//     temp = cupy.zeros(lenparents, dtype=toptr.dtype)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_countnonzero_a", cupy.dtype(toptr.dtype).type, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, temp, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_countnonzero_b", cupy.dtype(toptr.dtype).type, cupy.dtype(fromptr.dtype).type, parents.dtype]))((grid_size,), block, (toptr, fromptr, parents, lenparents, outlength, temp, invocation_index, err_code))
// out["awkward_reduce_countnonzero_a", {dtype_specializations}] = None
// out["awkward_reduce_countnonzero_b", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_countnonzero_a(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < outlength) {
      toptr[thread_id] = 0;
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_countnonzero_b(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    T* temp,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t idx = threadIdx.x;
    int64_t thread_id = blockIdx.x * blockDim.x + idx;

    if (thread_id < lenparents) {
      temp[thread_id] = (fromptr[thread_id] != 0) ? 1 : 0;
    }
    __syncthreads();

    if (thread_id < lenparents) {
      for (int64_t stride = 1; stride < blockDim.x; stride *= 2) {
        int64_t val = 0;
        if (idx >= stride && thread_id < lenparents && parents[thread_id] == parents[thread_id - stride]) {
          val = temp[thread_id - stride];
        }
        __syncthreads();
        temp[thread_id] += val;
        __syncthreads();
      }

      int64_t parent = parents[thread_id];
      if (idx == blockDim.x - 1 || thread_id == lenparents - 1 || parents[thread_id] != parents[thread_id + 1]) {
        atomicAdd(&toptr[parent], temp[thread_id]);
      }
    }
  }
}
